
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++) 
		c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; 
}

//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
	printf("\n");
}
	int main(void) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size); fill_array(b);
	c = (int *)malloc(size);

	// Alloc space for device copies of vector (a, b, c)
	hipMalloc((void **)&d_a, size); 
	hipMalloc((void **)&d_b, size); 
	hipMalloc((void **)&d_c, size); 

	// Copy from host to device 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 

	//host_add(a,b,c);
	device_add<<<N,1>>>(d_a, d_b, d_c); 

	// Copy result back to host 
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); 

	print_output(a,b,c);
	free(a); free(b); free(c);

	// Free GPU memory 
	hipFree(d_a); hipFree(d_b); hipFree(d_c); 

	return 0;
}
